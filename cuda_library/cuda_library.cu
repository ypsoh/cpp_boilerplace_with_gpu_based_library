#include "hip/hip_runtime.h"
// cuda_library.cu
#include <iostream>
#include <hip/hip_runtime.h>

#include "cuda_library.h"

// CUDA kernel to initialize data
__global__ void initializeDataKernel(float *data, int numElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements) {
        data[idx] = static_cast<float>(idx);
    }
}

// Function to initialize data using CUDA
void initializeData(float *data, int numElements) {
    // Allocate device memory
    float *d_data;
    hipMalloc((void**)&d_data, numElements * sizeof(float));

    // Launch the CUDA kernel to initialize data
    int threadsPerBlock = 256;
    int numBlocks = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    initializeDataKernel<<<numBlocks, threadsPerBlock>>>(d_data, numElements);

    // Copy data from device to host
    hipMemcpy(data, d_data, numElements * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_data);
}